#include "hip/hip_runtime.h"
#include <>
#include "ReconstructorCUDA.cuh"
#include "FileReaderCUDA.cuh"
namespace SLS
{

ReconstructorCUDA :: ReconstructorCUDA(const size_t projX, const size_t projY): 
    Reconstructor()
{
    projector_ = new Projector(projX, projY);
}
ReconstructorCUDA::~ReconstructorCUDA(){
    delete projector_;
}
void ReconstructorCUDA::addCamera(ImageFileProcessor* cam)
{
    cameras_.push_back(cam);
}

PointCloud ReconstructorCUDA::reconstruct(const std::vector<Buckets>&)
{
    return reconstruct();
}
PointCloud ReconstructorCUDA::reconstruct()
{
    // For each camera, supports only two cameras now.
    GPUBuckets buckets[2] =
    {
        GPUBuckets( projector_->getNumPixels(),110),
        GPUBuckets( projector_->getNumPixels(),110)
    };
    
    /**** Profile *****/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /**/

    // Color frames
    std::vector<uchar*> colors_d_;
    colors_d_.resize(cameras_.size());

    // For each camera
    for(size_t camIdx = 0; camIdx < cameras_.size(); camIdx++)
    {
        // Cast camera to CUDA camera
        FileReaderCUDA* cam = (FileReaderCUDA*)cameras_[camIdx];
        LOG::writeLog("Generating reconstruction bucket for \"%s\" ... \n", cam->getName().c_str());
        cam->computeShadowsAndThresholds();    // TODO: Put this part to GPU too.
        size_t x=0,y=0,xTimesY=0;
        cam->getResolution(x,y);
        xTimesY=x*y;

        // Load color images to GPU memory
        gpuErrchk( hipMalloc((void**)&(colors_d_[camIdx]), 
                    sizeof(uchar)* xTimesY * 3)); // num_pixel * (r,g,b)
        gpuErrchk( hipMemcpy( colors_d_[camIdx], cam->getColorFrame().data,
                    sizeof(uchar) * xTimesY * 3, hipMemcpyHostToDevice));

        // Skip first two frames
        cam->getNextFrame();
        cam->getNextFrame();
        // Load all images into GPU memory
        uchar *images_d=nullptr;

        // Allocate image memory on GPU with (number of pixel) * (number of pattern image + number of reversed pattern image)
        gpuErrchk(hipMalloc((void**)&images_d, sizeof(uchar)*xTimesY*projector_->getRequiredNumFrames()*2));

        // Initialize binary sequence for each pixel. Here we have xTimesY pixels and 
        // each pixel has `projector_->getRequiredNumFrames()` bits.
        Dynamic_Bitset_Array bitsetArray(xTimesY, projector_->getRequiredNumFrames());

        // Preparing data
        // Copy images to GPU
        for (size_t i=0; i<projector_->getRequiredNumFrames(); i++)
        {
            auto frm = cam->getNextFrame();
            auto invFrm = cam->getNextFrame();
            assert(frm.isContinuous() && invFrm.isContinuous());
            gpuErrchk( hipMemcpy( &images_d[xTimesY*2*i], frm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
            gpuErrchk( hipMemcpy( &images_d[xTimesY*(2*i+1)], invFrm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
        }

        // Generate bit array for all pixels from image sequence.
        Kernel::genPatternArray<<<200,200>>> (
                images_d, 
                projector_->getRequiredNumFrames(),
                xTimesY,
                cam->getWhiteThreshold(0),
                cam->getMask()->getGPUOBJ(),
                bitsetArray.getGPUOBJ()
                );
        // Check for errors
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipFree(images_d)); // Release the heavy image array

        // Insert pixels into bucket
        Kernel::buildBuckets<<<200, 200>>> (
             cam->getMask()->getGPUOBJ(),
             bitsetArray.getGPUOBJ(),
             xTimesY,
             vec2(projector_->getWidth(), projector_->getHeight()),
             buckets[camIdx].getGPUOBJ()
            );
        gpuErrchk(hipPeekAtLastError());
    }


    // some hacks down there, need to be refactored
    // Export point cloud in (x, y, z, r, g, b) 
    auto camera0 = (FileReaderCUDA*)(cameras_[0]);
    auto camera1 = (FileReaderCUDA*)(cameras_[1]);
    float* pointCloud_d = nullptr;  // Point cloud on device with x,y,z,r,g,b.
    size_t resX, resY;
    camera0->getResolution(resX, resY);

    gpuErrchk ( hipMalloc((void**)&pointCloud_d, buckets[0].getNumBKTs()*sizeof(float)*6));

    // Reconstructing point cloud
    LOG::writeLog("Reconstructing point cloud ...\n");
    Kernel::getPointCloud2Cam<<<200,200>>>(
            buckets[0].getGPUOBJ(),
            camera0->getDeviceCamMat(),
            camera0->getDeviceDistMat(),
            camera0->getDeviceCamTransMat(),
            colors_d_[0],

            buckets[1].getGPUOBJ(),
            camera1->getDeviceCamMat(),
            camera1->getDeviceDistMat(),
            camera1->getDeviceCamTransMat(),
            colors_d_[1],

            resX,resY,
            pointCloud_d
            );
    gpuErrchk(hipPeekAtLastError());

    PointCloud res(buckets[0].getNumBKTs());
    gpuErrchk(hipMemcpy(  res.getBuffer().data(), pointCloud_d, buckets[0].getNumBKTs()*sizeof(float)*6, hipMemcpyDeviceToHost));

    /**** Profile *****/
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    LOG::writeLog("GPU Time : %fms\n", milliseconds);
    /*****/

    gpuErrchk( hipFree(pointCloud_d));
    
    // Free color image device pointers
    for (const auto &ptr : colors_d_)
        gpuErrchk( hipFree(ptr));

    LOG::writeLog("Done\n");
    return res;
}

namespace Kernel{
// Kernels 

__global__ void genPatternArray(
        const uchar * imgs,
        size_t numImgs,
        size_t XtimesY,
        uchar whiteThreshold,
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)   // For each pixel
    {
        for (size_t i = 0; i<numImgs; i++)
        {
            if (!mask.getBit(0, idx)) 
            {
                // set the bit to black, unnecessary
                for (size_t j=0; j<numImgs; j++)
                    patterns.clearBit(j, idx);
                continue;
            }
            uchar pixel = imgs[ idx + XtimesY*(2*i)];
            uchar invPixel = imgs[ idx + XtimesY*(2*i+1)];
            if (invPixel > pixel && invPixel-pixel >= whiteThreshold)
                patterns.clearBit(numImgs-1-i, idx);
            else if (pixel > invPixel && pixel-invPixel > whiteThreshold)
                patterns.setBit(numImgs-1-i, idx);
            else
                mask.clearBit(0, idx);
        }
        idx += stride;
    }
}


// Insert image pixel indices into buckets
__global__ void buildBuckets(
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns,
        size_t XtimesY,
        vec2 projectorResolution,

        GPUBucketsObj bkts
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)   // For each pixel
    {
        glm::uvec2 bkt2v = patterns.to_uint_gray(idx);
        if (bkt2v.x < projectorResolution.x && bkt2v.y < projectorResolution.y && mask.getBit(0, idx))
            bkts.add2Bucket(idx, bkt2v.x+bkt2v.y*projectorResolution.x);
        idx += stride;
    }
}


// 
// Reconstruct point cloud from two cameras
// The inputs are the bucket, bit pattern and camera matrices.
// The output is written to float* pointCloud
//

__global__ void getPointCloud2Cam(
        GPUBucketsObj buckets0,
        float *camMat0,
        float *distMat0,
        float *camTransMat0,
        uchar* color0,

        GPUBucketsObj buckets1,
        float *camMat1,
        float *distMat1,
        float *camTransMat1,
        uchar* color1,

        uint camResX,
        uint camResY,

        float* pointCloud
        )
{
    // Each thread takes care of one projector pixel
    // i.e. a bucket
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < buckets0.NUM_BKTS_)   // For each pixel
    {
        if ( buckets0.count_[idx] == 0 || buckets1.count_[idx] == 0) 
        {
            // If there's no corresponding pixel
            // Set the point cloud to empty
            for(size_t i=0; i<6; i++)
                pointCloud[6 * idx + i] = 0.0;
        }
        else
        {
            //Undistorted pixels
            float minDist = 99999.0;
            uint minIdx0 = 0;
            uint minIdx1 = 0;
            float minMidPoint[4];

            float avgPoint[4];
            memset(avgPoint, 0, sizeof(float)*4);
            uint ptCount = 0;

            for (uint i=0; i<buckets0.count_[idx]; i++)
                for (uint j=0; j<buckets1.count_[idx]; j++)
                {

                    float undistorted0[2];
                    float undistorted1[2];

                    //Pick the first pixel in both buckets to test
                    undistortPixel(
                            buckets0.data_[idx*buckets0.MAX_CNT_PER_BKT_+i],
                            camResX, camResY,
                            camMat0, distMat0,
                            undistorted0);
                    undistortPixel(
                            buckets1.data_[idx*buckets1.MAX_CNT_PER_BKT_+j],
                            camResX, camResY,
                            camMat1, distMat1,
                            undistorted1);

                    float origin0[4];
                    float origin1[4];
                    float dir0[4];
                    float dir1[4];

                    getRay(undistorted0, camMat0, camTransMat0, 
                            origin0, dir0);
                    getRay(undistorted1, camMat1, camTransMat1, 
                            origin1, dir1);

                    float midPoint[4];

                    auto dist = getMidPoint(
                            origin0, dir0, origin1, dir1,
                            midPoint);
                    avgPoint[0] += midPoint[0];
                    avgPoint[1] += midPoint[1];
                    avgPoint[2] += midPoint[2];
                    avgPoint[3] += midPoint[3];
                    ptCount++;
                    if (dist < minDist)
                    {
                        minIdx0 = buckets0.data_[idx*buckets0.MAX_CNT_PER_BKT_+i];
                        minIdx1 = buckets1.data_[idx*buckets1.MAX_CNT_PER_BKT_+j];
                        minDist = dist;
                        memcpy (minMidPoint, midPoint, sizeof(float)*4);
                    }
                }
            avgPoint[0] /= (float)ptCount;
            avgPoint[1] /= (float)ptCount;
            avgPoint[2] /= (float)ptCount;
            avgPoint[3] = 1.0;
            float color[3] = {0.0, 0.0, 0.0};
            // OpenCV BGR to RGB
            color[2] = float(color0[minIdx0*3] + color1[minIdx1*3])/2;
            color[1] = float(color0[minIdx0*3+1] + color1[minIdx1*3+1])/2;
            color[0] = float(color0[minIdx0*3+2] + color1[minIdx1*3+2])/2;
            memcpy ( &pointCloud[6*idx], avgPoint, sizeof(float)*3);
            memcpy ( &pointCloud[6*idx+3], color, sizeof(float)*3);
        }
        idx += stride;
    }
}

} // namespace Kernel
} // namespace SLS
